#include "hip/hip_runtime.h"
__global__ void MatrixMulBlocksKernel(float *Md, float *Nd, float *Pd, int Width) {

	// Calculate the row index of the Pd element and M
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	// Calculate the column index of Pd and N
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	float Pvalue = 0;

	for (int k = 0; k < Width; ++k)
		Pvalue += Md[Row * Width + k] * Nd[k * Width + Col];

	Pd[Row * Width + Col] = Pvalue;
}

